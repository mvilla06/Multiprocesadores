
#include <hip/hip_runtime.h>
#include <time.h>
#include <stdio.h>

__global__ void kernel(const double * A, const double * B, double * C, int rows, int cols, int k){
  //  int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    //int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(i>=rows || j>=cols)
        return;
    
    
    for(int x=0; x<k; x++)
       
        C[i * cols + j] += A[i * k + x] * B[j * k + x]; 

}

extern "C" long int kernel_wrapper(const double * A, const double * B, double * C, int rowA, int colA, int rowB, int colB){
    time_t start, end;
    double * d_A;
    double * d_B;
    double * d_C;
    hipError_t error;
    
    error = hipMalloc((void**)&d_A, rowA * colA * sizeof(double));
    error = hipMalloc((void**)&d_B, rowB * colB * sizeof(double));
    error = hipMalloc((void**)&d_C, rowA * colB * sizeof(double));
    
    if(error){
        printf("Error reservando memoria GPU.\n");
        return 0;
    }
    hipMemset(d_C, 0, rowA * colB * sizeof(double));
    error = hipMemcpy((void*)d_A, (void*)A, rowA * colA * sizeof(double), hipMemcpyHostToDevice);
    
    error = hipMemcpy((void*)d_B, (void*)B, rowB * colB * sizeof(double), hipMemcpyHostToDevice);
    

    if(error){
        printf("Error copiando memoria a GPU.\n");
        return 0;
    }
    //Grid of 16x16 blocks
    int blocksX = (colB + 15)/16;
    int blocksY = (rowA + 15)/16;
    
    dim3 grid(blocksX, blocksY, 1);
    dim3 block(16, 16, 1);
    start = clock();
    kernel<<<grid, block>>>(d_A, d_B, d_C, rowA, colB, colA);
    end = clock();
    hipMemcpy(C, d_C, rowA * colB * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return (long int)end-start;
}