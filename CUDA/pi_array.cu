#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>

#define STEPS 2000000000
#define BLOCKS 100
#define THREADS 100

int threadidx;
double pi = 0;
clock_t start, end;
// Kernel
__global__ void pi_calculation(double* sum, int nsteps, double base, int nthreads, int nblocks)
{
int i;
double x;
int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate index for each thread
for (i = idx; i < nsteps; i += nthreads * nblocks)
{
x = (i + 0.5) * base;
sum[idx] += 4.0 / (1.0 + x * x); //Save result to device memory
}
}

int main(void)
{
dim3 dimGrid(BLOCKS, 1, 1); // Grid dimensions
dim3 dimBlock(THREADS, 1, 1); // Block dimensions
double* h_sum, * d_sum; // Pointer to host & device arrays
double base = 1.0 / STEPS; // base size
size_t size = BLOCKS * THREADS * sizeof(double); //Array memory size

//Memory allocation
h_sum = (double*)malloc(size); // Allocate array on host
hipMalloc((void**)&d_sum, size); // Allocate array on device
// Initialize array in device to 0
hipMemset(d_sum, 0, size);
start = clock();
// Launch Kernel
pi_calculation << <dimGrid, dimBlock >> > (d_sum, STEPS, base, THREADS, BLOCKS);

// Sync
hipDeviceSynchronize();

// Copy results from device to host
hipMemcpy(h_sum, d_sum, size, hipMemcpyDeviceToHost);

// Do the final reduction.
for (threadidx = 0; threadidx < THREADS * BLOCKS; threadidx++)
pi += h_sum[threadidx];

// Multiply by base
pi *= base;
end = clock();
// Output Results
printf("PI = %.10f\n (%ld)", pi, end-start);

// Cleanup
free(h_sum);
hipFree(d_sum);

return 0;
}