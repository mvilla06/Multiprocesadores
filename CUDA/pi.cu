
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#define NUM_INTERVALS 1000000
#define THREADS_PER_BLOCK 1000
#define BLOCKS 10

double baseIntervalo = 1.0/NUM_INTERVALS;

__global__ void piFunc(double * acum){
    int t = threadIdx.x;
    int threads = blockDim.x;
    
    acum[t] = 0;
    
    int intervals_per_thread = NUM_INTERVALS/(BLOCKS*THREADS_PER_BLOCK);
    double baseIntervalo = 1.0/NUM_INTERVALS;
    
    
    for(int i = (threads*blockIdx.x + t)*intervals_per_thread; i<(threads*blockIdx.x + t)*intervals_per_thread + intervals_per_thread; i++){
        
        double x = (i+0.5)*baseIntervalo;
        double fdx = 4 / (1 + x * x);
        acum[threads*blockIdx.x + t] += fdx;
    }

}

int main(){
    clock_t start, end;
    double h_pi[BLOCKS*THREADS_PER_BLOCK];
    double * d_pi;
    double pi = 0;

    hipMalloc(&d_pi, BLOCKS*THREADS_PER_BLOCK*sizeof(double));


    start = clock();
    piFunc<<<BLOCKS, THREADS_PER_BLOCK>>>(d_pi);
    hipMemcpy(h_pi, d_pi, BLOCKS*THREADS_PER_BLOCK*sizeof(double), hipMemcpyDeviceToHost);
    
    for(int i=0; i<BLOCKS*THREADS_PER_BLOCK; i++)
        //printf("%f", h_pi[i]);
        pi+=h_pi[i];

    pi *= baseIntervalo;
    end = clock();
    printf("Result = %20.18lf (%ld)\n", pi, end - start);

    hipFree(d_pi);


    return 0;
}