#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>

#define STEPS 2000000000
#define BLOCKS 100
#define THREADS 100

int threadidx;
double pi = 0;
clock_t start, end;

__device__ float pi_global;
// Kernel
__global__ void pi_calculation( int nsteps, double base, int nthreads, int nblocks)
{
    float accum =0;
int i;
float x;
int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate index for each thread
for (i = idx; i < nsteps; i += nthreads * nblocks)
{
x = (i + 0.5) * base;
accum += 4.0 / (1.0 + x * x);

//sum[idx] += 4.0 / (1.0 + x * x); //Save result to device memory
}
atomicAdd(&pi_global, accum);
}

int main(void)
{
dim3 dimGrid(BLOCKS, 1, 1); // Grid dimensions
dim3 dimBlock(THREADS, 1, 1); // Block dimensions

double base = 1.0 / STEPS; // base size
float pi_result = 0;



start = clock();
// Launch Kernel
pi_calculation << <dimGrid, dimBlock >> > (STEPS, base, THREADS, BLOCKS);

// Sync
hipDeviceSynchronize();




hipMemcpyFromSymbol(&pi_result, HIP_SYMBOL(pi_global), sizeof(float));
pi_result *= base;
end = clock();
// Output Results
printf("PI = %.10f\n (%ld)", pi_result, end-start);


return 0;
}